
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <time.h>


__global__
void calculate_neighbors(int sideLength, int *b, int *nb){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < sideLength*sideLength; i+=stride){
        nb[i] = 0;

        int y = i/sideLength;
        int x = i%sideLength;

        for(int dy = -1; dy <= 1; dy++){
            int ny = (y + dy + sideLength)%sideLength;

            for(int dx = -1; dx <= 1; dx++){
                if(!dx && !dy) continue;
                int nx = (x + dx + sideLength)%sideLength;

                nb[i] += b[ny*sideLength + nx];
            }
        }
    }
}

__global__
void iterate_boards(int sideLength, int *b1, int *b2, int *nb){
    //determine stride
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < sideLength*sideLength; i+=stride){
        if(b1[i]){
            if(nb[i] < 2 || nb[i] > 3){
                b2[i] = 0;
            } else b2[i] = 1;
        } else {
            if(nb[i] == 3){
                b2[i] = 1;
            } else {
                b2[i] = 0;
            }
        }
    }
}

void swap(int* &x, int* &y){
    int* cup = x;
    x = y;
    y = cup;
}

int main(int argc, char* argv[]){
    int sideLength = 1<<10;
    int iterations = 1<<9;
    double p = 0.5;

    //allocate unified memory -- accessible by both CPU and GPU
    int *board_1, *board_2, *neighbors_board;
    hipMallocManaged(&board_1, sideLength*sideLength);
    hipMallocManaged(&board_2, sideLength*sideLength);
    hipMallocManaged(&neighbors_board, sideLength*sideLength);


    //initialize random board
    srand(1);
    for(int i = 0; i < sideLength; i++){
        for(int j = 0; j < sideLength; j++){
            board_1[i* sideLength + j] = ((((double) rand())/RAND_MAX) <= p);

        }
    }

    int blockSize = 256; //blockSize must be a power of 2
    int numBlocks = (sideLength*sideLength + blockSize - 1) / blockSize; //first let's separate cell by cell.


    for(int i = 0; i < iterations; i++){
        calculate_neighbors<<<numBlocks,blockSize>>>(sideLength,board_1,neighbors_board);
        hipDeviceSynchronize();
        iterate_boards<<<numBlocks,blockSize>>>(sideLength,board_1,board_2,neighbors_board);
        hipDeviceSynchronize();
        swap(board_1,board_2);
    }

    //free allocated memory
    hipFree(board_1);
    hipFree(board_2);
    hipFree(neighbors_board);

    return 0;
}